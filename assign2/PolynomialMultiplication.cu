#include "hip/hip_runtime.h"
#include <unistd.h>
#include <stdio.h>
#include <time.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdarg.h>
#include <string.h>
#include <hip/hip_runtime.h>

#define GPU_DEVICE 0
#define EPSILON 1e-6

// CUDA Parallel Polynomial Multiplication
__global__ void polynomialMultiplication(int* A, int* B, int* C, int n) {
    extern __shared__ int shared[];
    int* A_shared = shared;
    int* B_shared = shared + blockDim.x;

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int p_start = max(0, n - idx);
    int p_end = min(n, 2 * n - idx);

    int result = 0;

    // Load data into shared memory
    if (p_start <= p_end) {
        for (int t = p_start; t <= p_end; t += blockDim.x) {
            int t_idx = t + tid;
            A_shared[tid] = (t_idx <= n) ? A[t_idx] : 0;
            B_shared[tid] = (idx - n + t_idx <= n) ? B[idx - n + t_idx] : 0;
            __syncthreads();

            // Perform multiplication and reduction
            for (int i = 0; i < min(blockDim.x, p_end - t + 1); i++) {
                result += A_shared[i] * B_shared[blockDim.x - 1 - i];
            }
            __syncthreads();
        }
    }

    // Write result to global memory
    if (idx <= 2 * n) {
        C[idx] = result;
    }
}

// Serial Polynomial Multiplication (Same as above just only on CPU)
void polynomialMultiplicationSerial(int* A, int* B, int* C, int n) {
    for (int idx = 0; idx <= 2 * n; idx++) {
        int p_start = max(0, n - idx);
        int p_end = min(n, 2 * n - idx);
        int result = 0;
        for (int t = p_start; t <= p_end; t++) {
            result += A[t] * B[idx - n + t];
        }
        C[idx] = result;
    }
}


bool verifyResults(int* C1, int* C2, int N) {
    for (int i = 0; i < 2 * N; i++) {
        if (fabs(C1[i] - C2[i]) > EPSILON) {
            return false;
        }
    }
    return true;
}

int main() {
    int B_values[] = {32, 64, 128, 256, 512};
    int N_values[] = {(int)pow(2, 14), (int)pow(2, 16)};

    printf("\\begin{table}[ht]\n");
    printf("\\centering\n");
    printf("\\begin{tabular}{|c|c|c|c|c|}\n");
    printf("\\hline\n");
    printf("N & B & GPU Time (ms) & CPU Time (ms) & Speedup \\\\\n");
    printf("\\hline\n");

    // Run the kernel for different values of B and N and print out GPU vs CPU time
    for (int j = 0; j < 2; j++) {
        int N = N_values[j];
        for (int i = 0; i < 5; i++) {
            int B = B_values[i];
            int* A = (int*)malloc((N + 1) * sizeof(int));
            int* Br = (int*)malloc((N + 1) * sizeof(int));
            int* C = (int*)malloc((2 * N + 1) * sizeof(int));
            int* C_serial = (int*)malloc((2 * N + 1) * sizeof(int));
            
            // Random values from {-1, 0, 1}
            for (int i = 0; i < N + 1; i++) {
                A[i] = rand() % 3 - 1;
                Br[i] = rand() % 3 - 1;
            }

            int* d_A;
            int* d_B;
            int* d_C;
            hipMalloc(&d_A, (N + 1) * sizeof(int));
            hipMalloc(&d_B, (N + 1) * sizeof(int));
            hipMalloc(&d_C, (2 * N + 1) * sizeof(int));

            hipMemcpy(d_A, A, (N + 1) * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(d_B, Br, (N + 1) * sizeof(int), hipMemcpyHostToDevice);

            struct timeval start, end;
            gettimeofday(&start, NULL);
            polynomialMultiplication<<<(2 * N + B - 1) / B, B>>>(d_A, d_B, d_C, N);
            hipDeviceSynchronize();
            gettimeofday(&end, NULL);
            float gpu_time = (end.tv_sec - start.tv_sec) * 1e6 * 1000;
            gpu_time = (gpu_time + (end.tv_usec - start.tv_usec)) * 1e-6;

            hipMemcpy(C, d_C, (2 * N + 1) * sizeof(int), hipMemcpyDeviceToHost);

            gettimeofday(&start, NULL);
            polynomialMultiplicationSerial(A, Br, C_serial, N);
            gettimeofday(&end, NULL);
            float time = (end.tv_sec - start.tv_sec) * 1e6 * 1000;
            time = (time + (end.tv_usec - start.tv_usec)) * 1e-6;
            bool valid = verifyResults(C, C_serial, N);
            if (valid) {
                printf("%d & %d & %.1e & %.1e & %.1e \\\\\n", N, B, gpu_time, time, time / gpu_time);
            }
            else {
                printf("%d & %d & %.1e & %.1e & %.1e \\\\\n", N, B, gpu_time, time, time / gpu_time);
            }

            free(A);
            free(Br);
            free(C);
            free(C_serial);
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
        }
    }

    printf("\\hline\n");
    printf("\\end{tabular}\n");
    printf("\\caption{GPU vs CPU Time Comparison}\n");
    printf("\\label{tab:comparison}\n");
    printf("\\end{table}\n");
}